#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include "hip/hip_fp16.h"

#define TILE_WIDTH 16
__global__ void conv_forward_kernel(half *output, const half *input, const half *mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int W_tile = ceil(1.0 * W_out / TILE_WIDTH);
    int H_tile = ceil(1.0 * H_out / TILE_WIDTH);
    
    int h = (bz / W_tile) * TILE_WIDTH + ty;
    int w = (bz % W_tile) * TILE_WIDTH + tx;

    half acc = 0.0;

    for (int c = 0; c < C; c++) {
        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                if (h * S + p < H && w * S + q < W) {
                    acc = __hadd(__hmul(in_4d(bx, c, h*S+p, w*S+q), mask_4d(by, c, p, q)), acc);
                }
            }
        }
    }
    if (h < H_out && w < W_out) {
        out_4d(bx, by, h, w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__global__ void float2halfArray(const float *input, half *output, const int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) {
        output[i] = __float2half(input[i]);
    }
}

__global__ void half2floatArray(half *input, float *output, const int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) {
        output[i] = __half2float(input[i]);
    }
}
	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory for device_output, device_input, and device_mask
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    hipMalloc((void**) device_output_ptr, B * M * H_out * W_out * sizeof(float));
    hipMalloc((void**) device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void**) device_mask_ptr, M * C * K * K * sizeof(float));

    // Copy data from host to device
    hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
   
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    int W_tile = ceil(1.0 * W_out / TILE_WIDTH);
    int H_tile = ceil(1.0 * H_out / TILE_WIDTH);
    int Z = W_tile * H_tile;


    half *device_input_half;
    half *device_mask_half;
    half *device_output_half;
    hipMalloc((void**)&device_input_half, sizeof(half) * B * C * H * W);
    hipMalloc((void**)&device_mask_half, sizeof(half) * M * C * K * K);
    hipMalloc((void**)&device_output_half, sizeof(half) * B * M * H_out * W_out);

    dim3 dimGrid(16, 1, 1);
    dim3 dimBlock(512, 1, 1);
    float2halfArray<<<dimGrid, dimBlock>>>(device_input, device_input_half, B * C * H * W);
    hipDeviceSynchronize();
    float2halfArray<<<dimGrid, dimBlock>>>(device_mask, device_mask_half, M * C * K * K);
    hipDeviceSynchronize();

    dim3 dimGrid2(B, M, Z);
    dim3 dimBlock2(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<dimGrid2, dimBlock2>>>(device_output_half, device_input_half, device_mask_half, B, M, C, H, W, K, S);
    hipDeviceSynchronize();

    dim3 dimGrid3(16, 1, 1);
    dim3 dimBlock3(1024, 1, 1);
    half2floatArray<<<dimGrid3, dimBlock3>>>(device_output_half, device_output, B * M * H_out * W_out);
    hipDeviceSynchronize();

    hipFree(device_input_half);
    hipFree(device_mask_half);
    hipFree(device_output_half);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    hipMemcpy(host_output, device_output, B * M * W_out * H_out * sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
